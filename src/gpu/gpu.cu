#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "util/common.h"

#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if( r2 > cutoff*cutoff )
        return;
    //r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
    double r = sqrt( r2 );

    //
    //  very simple short-range repulsive force
    //
    double coef = ( 1 - cutoff / r ) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;

}

__global__ void bin_particles_gpu (particle_t *particles, int n, particle_t ** bins, 
                                   int * num_particles_in_bins, int n_bins_per_side,
                                   int n_max_particles_per_bin) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;

    int bin_x_id = floor(particles[tid].x / bin_size);
    int bin_y_id = floor(particles[tid].y / bin_size);
    int bin_id = bin_x_id * n_bins_per_side + bin_y_id;

    int index_in_bin = atomicAdd(num_particles_in_bins+bin_id, 1);
    bins[n_max_particles_per_bin * bin_id + index_in_bin] = particles + tid;
}

__device__ void apply_force_particle_bin_gpu (particle_t &particle, int bin_id, particle_t ** bins, 
                                              int * num_particles_in_bins, int n_bins_per_side, 
                                              int n_max_particles_per_bin) {
    int bin_x_id = bin_id / n_bins_per_side;
    int bin_y_id = bin_id % n_bins_per_side;

    particle.ax = particle.ay = 0;

    // apply force within bin
    for (int j = 0; j < num_particles_in_bins[bin_id]; j++) {
      apply_force_gpu(particle, *(bins[bin_id * n_max_particles_per_bin + j]));
    }


    // apply force from edge-neighboring bins
    if (bin_x_id != 0) {
      int neighbor_id = bin_id - n_bins_per_side;
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }
    if (bin_x_id != n_bins_per_side-1) {
      int neighbor_id = bin_id + n_bins_per_side;
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }
    if (bin_y_id != 0) {
      int neighbor_id = bin_id - 1;
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }
    if (bin_y_id != n_bins_per_side-1) {
      int neighbor_id = bin_id + 1;
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }

    // apply force from edge-neighboring bins
    if (bin_x_id != 0 && bin_y_id != 0) {
      int neighbor_id = bin_id - (n_bins_per_side + 1);
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }
    if (bin_x_id != 0 && bin_y_id != n_bins_per_side-1) {
      int neighbor_id = bin_id - (n_bins_per_side - 1);
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }
    if (bin_x_id != n_bins_per_side-1 && bin_y_id != 0) {
      int neighbor_id = bin_id + n_bins_per_side - 1;
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }
    if (bin_x_id != n_bins_per_side-1 && bin_y_id != n_bins_per_side-1) {
      int neighbor_id = bin_id + n_bins_per_side + 1;
      for (int j = 0; j < num_particles_in_bins[neighbor_id]; j++) {
        apply_force_gpu(particle, *(bins[neighbor_id * n_max_particles_per_bin + j]));
      }
    }


}

__global__ void compute_forces_bin_gpu (particle_t ** bins, int * num_particles_in_bins, int n_bins_per_side, 
                                        int n_max_particles_per_bin) {
    int bid = threadIdx.x + blockIdx.x * blockDim.x;
    int n_bins = n_bins_per_side * n_bins_per_side;
    if (bid >= n_bins) return;

    if (num_particles_in_bins[bid] == 0) return;

    for (int i = 0; i < num_particles_in_bins[bid]; i++) {
      apply_force_particle_bin_gpu( *(bins[bid * n_max_particles_per_bin + i]), bid, bins, num_particles_in_bins, n_bins_per_side, n_max_particles_per_bin);
    }
}

__global__ void compute_forces_gpu(particle_t * particles, int n)
{
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particles[tid].ax = particles[tid].ay = 0;
    for(int j = 0 ; j < n ; j++)
      apply_force_gpu(particles[tid], particles[j]);

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}



int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    int n_bins_per_side = ceil(size / bin_size);
    int n_bins = n_bins_per_side * n_bins_per_side;
    int n_max_particles_per_bin = 10;   // magic number

    particle_t **d_bins; 
    hipMalloc((void ***) &d_bins, n_bins * n_max_particles_per_bin * sizeof(particle_t *));

    int *num_particles_in_bins;
    hipMalloc((void **) &num_particles_in_bins, n_bins * sizeof(int *));

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int bin_blks = (n_bins + NUM_THREADS - 1) / NUM_THREADS;

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //
        hipMemset(num_particles_in_bins, 0, n_bins * sizeof(int));

        bin_particles_gpu <<< blks, NUM_THREADS >>> (d_particles, n, d_bins, num_particles_in_bins, n_bins_per_side, n_max_particles_per_bin);

        compute_forces_bin_gpu <<< bin_blks, NUM_THREADS >>> (d_bins, num_particles_in_bins, n_bins_per_side, n_max_particles_per_bin);
//        compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);
        
        //
        //  move particles
        //
        move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
