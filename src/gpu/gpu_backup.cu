#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "util/common.h"

#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__device__ void apply_force_particle_bin(particle_t &particle, int bin_id, particle_t **bins,
                                         int* num_particles_per_bin, int n_bins_per_side, int max_particles_per_bin)
{
  int bin_x_id = bin_id / n_bins_per_side;
  int bin_y_id = bin_id % n_bins_per_side;

  // reset acceleration
  particle.ax = particle.ay = 0;

  // apply force within bin
  for (int j = 0; j < num_particles_per_bin[bin_id]; j++) {
    apply_force_gpu(particle, *(bins[bin_id*max_particles_per_bin+j]));
  }

  // apply force from edge-neighboring bins
  if (bin_x_id != 0) {
    int neighbor_id = bin_id - n_bins_per_side;
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }
  if (bin_x_id != n_bins_per_side-1) {
    int neighbor_id = bin_id + n_bins_per_side;
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }
  if (bin_y_id != 0) {
    int neighbor_id = bin_id - 1;
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }
  if (bin_y_id != n_bins_per_side-1) {
    int neighbor_id = bin_id + 1;
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }

  // apply force from edge-neighboring bins
  if (bin_x_id != 0 && bin_y_id != 0) {
    int neighbor_id = bin_id - (n_bins_per_side + 1);
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }
  if (bin_x_id != 0 && bin_y_id != n_bins_per_side-1) {
    int neighbor_id = bin_id - (n_bins_per_side - 1);
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }
  if (bin_x_id != n_bins_per_side-1 && bin_y_id != 0) {
    int neighbor_id = bin_id + n_bins_per_side - 1;
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }
  if (bin_x_id != n_bins_per_side-1 && bin_y_id != n_bins_per_side-1) {
    int neighbor_id = bin_id + n_bins_per_side + 1;
    for (int j = 0; j < num_particles_per_bin[neighbor_id]; j++) {
      apply_force_gpu(particle, *(bins[neighbor_id*max_particles_per_bin+j]));
    }
  }
}

__global__ void bin_particles_gpu(particle_t **bins, int* num_particles_per_bin, int n_bins_per_side, 
                                  int max_particles_per_bin, particle_t *particles, int n) 
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= n) return;

  int bin_x_id = floor(particles[tid].x / bin_size);
  int bin_y_id = floor(particles[tid].y / bin_size);
  int bin_id = bin_x_id * n_bins_per_side + bin_y_id;
  int id_in_bin = atomicAdd(num_particles_per_bin+bin_id, 1);

  bins[bin_id * max_particles_per_bin + id_in_bin] = particles+tid;
}

__global__ void compute_forces_bin_gpu(particle_t **bins, int* num_particles_per_bin, int n_bins_per_side,
                                       int max_particles_per_bin)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int n_bins = n_bins_per_side * n_bins_per_side;
  if (tid >= n_bins) return;

  if (num_particles_per_bin[tid] == 0) return;

  for (int i = 0; i < num_particles_per_bin[tid]; i++) {
    apply_force_particle_bin(*(bins[tid*max_particles_per_bin+i]), tid, bins, num_particles_per_bin,
                             n_bins_per_side, max_particles_per_bin);
  }
}

__global__ void compute_forces_gpu(particle_t * particles, int n)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++)
    apply_force_gpu(particles[tid], particles[j]);

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
  //
  //  slightly simplified Velocity Verlet integration
  //  conserves energy better than explicit Euler method
  //
  p->vx += p->ax * dt;
  p->vy += p->ay * dt;
  p->x  += p->vx * dt;
  p->y  += p->vy * dt;

  //
  //  bounce from walls
  //
  while( p->x < 0 || p->x > size )
  {
      p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
      p->vx = -(p->vx);
  }
  while( p->y < 0 || p->y > size )
  {
      p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
      p->vy = -(p->vy);
  }

}



int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    particle_t **bins;
    int n_bins_per_side = ceil(size / bin_size);
    int n_bins = n_bins_per_side * n_bins_per_side;
    int max_num_particles_per_bin_per_side = 8;
    int max_num_particles_per_bin = (ceil(bin_size/cutoff)*max_num_particles_per_bin_per_side) * (ceil(bin_size/cutoff)*max_num_particles_per_bin_per_side);
    hipMalloc((void ***) &bins, max_num_particles_per_bin * n_bins * sizeof(particle_t *));

    int* num_particles_per_bin;
    hipMalloc((void **) &num_particles_per_bin, n_bins * sizeof(int));

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int bin_blks = (n_bins + NUM_THREADS - 1) / NUM_THREADS;

    for( int step = 0; step < NSTEPS; step++ )
    {
      //
      //  compute forces
      //
      hipMemset(num_particles_per_bin, 0, n_bins * sizeof(int));

      bin_particles_gpu <<<blks, NUM_THREADS>>> (bins, num_particles_per_bin, n_bins_per_side, 
                                                 max_num_particles_per_bin, d_particles, n);

      compute_forces_bin_gpu <<<bin_blks, NUM_THREADS>>> (bins, num_particles_per_bin, n_bins_per_side,
                                                          max_num_particles_per_bin);
//      compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);
        
      //
      //  move particles
      //
      move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
      
      //
      //  save if necessary
      //
      if( fsave && (step%SAVEFREQ) == 0 ) {
        // Copy the particles back to the CPU
        hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
        save( fsave, n, particles);
      }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
